#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include "element_wise.cuh"
#include <tensor.h>
#include <kernels/registry.h>

namespace rwkv {
namespace cuda {

Tensor layer_norm_op(const Tensor& x, const Tensor& weight, const Tensor& bias);

namespace {
// Equivalent Python code:
// ww = t_first + k
// p = torch.maximum(pp, ww)
// e1 = torch.exp(pp - p)
// e2 = torch.exp(ww - p)
// wkv = ((e1 * aa + e2 * v) / (e1 * bb + e2)).to(dtype=x.dtype)
// ww = t_decay + pp
// p = torch.maximum(ww, k)
// e1 = torch.exp(ww - p)
// e2 = torch.exp(k - p)
// t1 = e1 * aa + e2 * v
// t2 = e1 * bb + e2
// r = r * wkv
// return t1, t2, p, r
struct WkvForwardOne {
  const float *t_first;
  const float *k;
  const float *pp;
  const float *aa;
  const float *bb;
  const float *t_decay;
  const float *v;
  /* out */ float *t1;
  /* out */ float *t2;
  /* out */ float *p;
  /* in & out */ half *r;

  __device__ void operator()(int i) const {
    float ww = t_first[i] + k[i];
    float pp_ = pp[i];
    float p_ = (pp_ > ww) ? pp_ : ww;
    float e1 = expf(pp_ - p_);
    float e2 = expf(ww - p_);
    float aa_ = aa[i];
    float bb_ = bb[i];
    float v_ = v[i];
    r[i] = __hmul(r[i], __float2half(((e1 * aa_ + e2 * v_) / (e1 * bb_ + e2))));
    ww = t_decay[i] + pp_;
    float k_ = k[i];
    p_ = (ww > k_) ? ww : k_;
    e1 = expf(ww - p_);
    e2 = expf(k_ - p_);
    t1[i] = e1 * aa_ + e2 * v_;
    t2[i] = e1 * bb_ + e2;
    p[i] = p_;
  }
};

/*
   Equivalent Python code:
   kx = xx * k_mix + sx * (1 - k_mix)
   vx = xx * v_mix + sx * (1 - v_mix)
   rx = xx * r_mix + sx * (1 - r_mix)
*/

struct Mix {
  const half *xx;
  const half *sx;
  const half *k_mix;
  const half *v_mix;
  const half *r_mix;
  /* out */ half *kx;
  /* out */ half *vx;
  /* out */ half *rx;

  __device__ void operator()(int i) const {
    half xx_ = xx[i];
    half sx_ = sx[i];
    half k_mix_ = k_mix[i];
    half v_mix_ = v_mix[i];
    half r_mix_ = r_mix[i];
    kx[i] = __hadd(__hmul(xx_, k_mix_),
                   __hmul(sx_, __hsub(__float2half(1), k_mix_)));
    vx[i] = __hadd(__hmul(xx_, v_mix_),
                   __hmul(sx_, __hsub(__float2half(1), v_mix_)));
    rx[i] = __hadd(__hmul(xx_, r_mix_),
                   __hmul(sx_, __hsub(__float2half(1), r_mix_)));
  }
};

struct InplaceSigmoid {
  __device__ __forceinline__ void operator()(int i) const {
    ptr[i] = __float2half(1.0 / (1.0 + exp(-__half2float(ptr[i]))));
  }
  half *ptr;
};

struct InplaceAdd {
  __device__ __forceinline__ void operator()(int i) const {
    y[i] = __hadd(x[i], y[i]);
  }
  half *y;
  const half *x;
};

} // namespace

void gemm_cublas_tensor(const Tensor &a, const Tensor &b, Tensor &c);

Tensor _ATT(const Tensor& x, const Tensor& ln_w, const Tensor& ln_b, const Tensor& sx, const Tensor& k_mix,
            const Tensor& v_mix, const Tensor& r_mix, const Tensor& kw,
            /* imm */ Tensor& kx, const Tensor& vw, /* imm */ Tensor& vx, const Tensor& rw,
            /* imm */ Tensor& rx, const Tensor& ow, const Tensor& t_first,
            /* imm */ Tensor& k, const Tensor& pp, const Tensor& ww, const Tensor& aa, const Tensor& bb,
            const Tensor& t_decay, /* imm */ Tensor& v, /* in & out */ Tensor& r,
            /* out */ Tensor& x_plus_out, /* out */ Tensor& t1,
            /* out */ Tensor& t2, /* out */ Tensor& p) {
  Tensor xx = cuda::layer_norm_op(x, ln_w, ln_b);
  element_wise(Mix{xx.data_ptr<half>(), sx.data_ptr<half>(),
                   k_mix.data_ptr<half>(), v_mix.data_ptr<half>(),
                   r_mix.data_ptr<half>(), kx.data_ptr<half>(),
                   vx.data_ptr<half>(), rx.data_ptr<half>()},
               x.numel());

  gemm_cublas_tensor(kx, kw, k);
  gemm_cublas_tensor(vx, vw, v);
  gemm_cublas_tensor(rx, rw, r);
  element_wise(InplaceSigmoid{r.data_ptr<half>()}, r.numel());

  element_wise(WkvForwardOne{t_first.data_ptr<float>(), k.data_ptr<float>(),
                             pp.data_ptr<float>(), aa.data_ptr<float>(),
                             bb.data_ptr<float>(), t_decay.data_ptr<float>(),
                             v.data_ptr<float>(), t1.data_ptr<float>(),
                             t2.data_ptr<float>(), p.data_ptr<float>(),
                             r.data_ptr<half>()},
               x.numel());

  gemm_cublas_tensor(r, ow, x_plus_out);
  element_wise(InplaceAdd{x_plus_out.data_ptr<half>(), x.data_ptr<half>()},
               x.numel());
  return xx;
}

inline std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor>
att(const Tensor& x, const Tensor& sx, const Tensor& aa, const Tensor& bb, const Tensor& pp, const Tensor& ln_w,
    const Tensor& ln_b, const Tensor& k_mix, const Tensor& v_mix, const Tensor& r_mix, const Tensor& t_decay,
    const Tensor& t_first, const Tensor& kw, const Tensor& vw, const Tensor& rw, const Tensor& ow) {

  // kx = torch.empty_like(x)
  // vx = torch.empty_like(x)
  // rx = torch.empty_like(x)
  //
  // k_t = torch.empty((kw.shape[0],), dtype=torch.float32, device=x.device)
  // v_t = torch.empty((vw.shape[0],), dtype=torch.float32, device=x.device)
  // r_t = torch.empty((rw.shape[0],), dtype=torch.float16, device=x.device)
  // x_plus_out_t = torch.empty_like(x)
  // t1_t = torch.empty_like(x, dtype=torch.float32)
  // t2_t = torch.empty_like(x, dtype=torch.float32)
  // p_t = torch.empty_like(x, dtype=torch.float32)
  //             xx = torch.ops.rwkv.att_one(x, ln_w, ln_b, sx, k_mix, v_mix,
  //             r_mix, kw, kx, vw, vx, rw, rx, ow, t_first, k_t, pp, ow, aa,
  //             bb, t_decay, v_t, r_t, x_plus_out_t, t1_t, t2_t, p_t)
  // return x_plus_out_t, xx, t1_t, t2_t, p_t

  auto kx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto vx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto rx = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto k = Tensor::Empty({kw.size(0)}, DType::kFloat32, x.device());
  auto v = Tensor::Empty({vw.size(0)}, DType::kFloat32, x.device());
  auto r = Tensor::Empty({rw.size(0)}, DType::kFloat16, x.device());
  auto x_plus_out = Tensor::Empty(x.sizes(), x.dtype(), x.device());
  auto t1 = Tensor::Empty(x.sizes(), DType::kFloat32, x.device());
  auto t2 = Tensor::Empty(x.sizes(), DType::kFloat32, x.device());
  auto p = Tensor::Empty(x.sizes(), DType::kFloat32, x.device());

  Tensor xx =
      _ATT(x, ln_w, ln_b, sx, k_mix, v_mix, r_mix, kw, kx, vw, vx, rw, rx, ow,
           t_first, k, pp, vw, aa, bb, t_decay, v, r, x_plus_out, t1, t2, p);
  return std::make_tuple(x_plus_out, xx, t1, t2, p);
}

KernelRegister att_reg("att", Device::kCUDA, att);

} // namespace cuda
} // namespace rwkv
